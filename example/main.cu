#include "hip/hip_runtime.h"

/* INCLUDES */

// Own
#include "field_element_cuda.cuh" // struct Field_Element

// Nvcc
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void test_field_element() {

    // Field Elements creation
    printf("Hello, World! From GPU\n");

}

int main() {

    // Tests about field element
    test_field_element<<< 1, 1 >>>();

    // Synchronize cuda threads
    hipDeviceSynchronize();

}