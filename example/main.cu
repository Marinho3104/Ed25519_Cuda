#include "hip/hip_runtime.h"

/* INCLUDES */

#include "pre_computed_values.cuh" 
#include "field_element.cuh"
#include "ed25519.cuh"
#include "sha512.cuh"

// Nvcc
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void test_operations() {

    unsigned char _private_key[ 32 ] { 0, 1, 2, 3, 4, 5 };
    unsigned char _public_key[ 32 ];

    printf( "Private key: \n");
    for ( int _ = 0; _ < 32; _++ ) printf("%02x", _private_key[ _ ] ); printf("\n");

    ed25519::cuda::generate_key_pair( _private_key, _public_key );

    printf( "Public key: \n");
    for ( int _ = 0; _ < 32; _++ ) printf("%02x", _public_key[ _ ] ); printf("\n");

    printf("-------------------------\n");

    unsigned char _text[] = "afghkdfbfbfbrgbertbtbbegnwvlwrvnelrvnwrvjrv5ougbneuioneuog3ungnu5uh33g389h8935g3v3j59vn23vn93un93n935n34nvun93n93nvu3nu93nu9v3nnv93vnuv3nuv3nuvnu3vun3g3rgjrjir32jr23838rv83rg8903v3rng0r3n8gn380rgnr3gn0r3g0jn3rgn03rn80g3v38n0v30n8v305n8g8jng3nv8n38nj0v308vb083v8n3vn083n80v30n8v30n8v3n08vn038n08afghkdfbfbfbrgbertbtbbegnwvlwrvnelrvnwrvjrv5ougbneuioneuog3ungnu5uh33g389h8935g3v3j59vn23vn93un93n935n34nvun93n93nvu3nu93nu9v3nnv93vnuv3nuv3nuvnu3vun3g3rgjrjir32jr23838rv83rg8903v3rng0r3n8gn380rgnr3gn0r3g0jn3rgn03rn80g3v38n0v30n8v305n8g8jng3nv8n38nj0v308vb083v8n3vn083n80v30n8v30n8v3n08vn038n08afghkdfbfbfbrgbertbtbbegnwvlwrvnelrvnwrvjrv5ougbneuioneuog3ungnu5uh33g389h8935g3v3j59vn23vn93un93n935n34nvun93n93nvu3nu93nu9v3nnv93vnuv3nuv3nuvnu3vun3g3rgjrjir32jr23838rv83rg8903v3rng0r3n8gn380rgnr3gn0r3g0jn3rgn03rn80g3v38n0v30n8v305n8g8jng3nv8n38nj0v308vb083v8n3vn083n80v30n8v30n8v3n08vn038n08";

    unsigned char _signaure[ 64 ];

    ed25519::cuda::sign( _private_key, _text, 882, _signaure );

    printf( "Signature: " );
    for ( int _ = 0; _ < 64; _++ ) printf("%02x", _signaure[ _ ] ); printf("\n");

    printf( "Verification: %d\n", ed25519::cuda::verify( _public_key, _signaure, _text, 882 ) );

}

int main() {

    // Run
    test_operations <<< 1, 1 >>>();

    hipDeviceSynchronize();

}