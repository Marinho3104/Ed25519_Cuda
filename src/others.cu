

#include "others.cuh"
#include <stdarg.h>


__device__ void ed25519::cuda::sha512_multiple_args( sha512::cuda::Sha512_Context* __ctx, uint32_t __max_size, uint32_t __count, ... ) {

    va_list _args; va_start( _args, __count );

    void* _message; hipMalloc( &_message, __max_size );
    uint32_t _current_arg_size; void* _current_arg;

    while ( __count -- ) {

        _current_arg_size = va_arg( _args, uint32_t );
        _current_arg = va_arg( _args, void* );

        memcpy( _message, _current_arg, _current_arg_size );

        _message = _message + _current_arg_size; 

    }

    _message = _message - __max_size; 

    // for ( uint64_t _ = 0; _ < __max_size; _ ++ ) {

    //     printf( "%02x", _message + _ );

    // } printf( "\n" );

    sha512::cuda::Sha512_hash( __ctx, _message, __max_size );

}
