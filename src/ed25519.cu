

#include "pre_computed_values.cuh"
#include "field_element.cuh"
#include "group_element.cuh"
#include "ed25519.cuh"
#include "others.cuh"
#include "sha512.cuh"

#include <stdio.h>


__device__ void ed25519::cuda::generate_key_pair( void* __private_key, void* __public_key ) {

    sha512::cuda::Sha512_Context _ctx; sha512::cuda::Sha512_init( &_ctx );

    sha512::cuda::Sha512_hash( &_ctx, __private_key, 32 ); sha512::cuda::Sha512_free( &_ctx );

    unsigned char* _seed = _ctx.hash;

    // Clear the lowest 3 bits of the first octet
    _seed[ 0 ] &= 0xf8;

    // Clear the highest bit of the last octet
    _seed[ 31 ] &= 0x3f;

    // Set the second highest bit of the last octet
    _seed[ 31 ] |= 0x40;

    group_element_ec _base_point_ec; set_extended_coordinates( &_base_point_ec, base_point );
    field_element _scalar; memcpy( _scalar, _seed, 32 );

    group_element_ec _result; scalar_multiplication( &_result, _base_point_ec, _scalar );

    encode( __public_key, _result );

}

__device__ void ed25519::cuda::sign( void* __private_key, void* __message, uint64_t __message_size, void* __signature ) {

    sha512::cuda::Sha512_Context* _ctx; hipMalloc( &_ctx, sizeof( sha512::cuda::Sha512_Context ) );
    group_element_ec* _base_point_ec; hipMalloc( &_base_point_ec, sizeof( group_element_ec ) );

    field_element* _s, *_r, *_k, *_S; 
    hipMalloc( &_s, sizeof( field_element ) ); zeralise( *_s );
    hipMalloc( &_r, sizeof( field_element ) ); zeralise( *_r );
    hipMalloc( &_k, sizeof( field_element ) ); zeralise( *_k );
    hipMalloc( &_S, sizeof( field_element ) ); zeralise( *_S );

    unsigned char* _public_key_A; hipMalloc( &_public_key_A, 32 );
    unsigned char* _prefix; hipMalloc( &_prefix, 32 );
    unsigned char* _R; hipMalloc( &_R, 32 );


    // Hash private key
    sha512::cuda::Sha512_init( _ctx ); sha512::cuda::Sha512_hash( _ctx, __private_key, 32 ); sha512::cuda::Sha512_free( _ctx );

    // Generate public key 
    set_extended_coordinates( _base_point_ec, base_point );
    memcpy( *_s, _ctx->hash, 32 ); ( ( unsigned char* ) *_s )[ 0 ] &= 0xf8; ( ( unsigned char* ) *_s )[ 31 ] &= 0x3f; ( ( unsigned char* ) *_s )[ 31 ] |= 0x40; 
    group_element_ec _pk_A_ec { { 0 }, { 0 }, { 0 }, { 0 } }; scalar_multiplication( &_pk_A_ec, *_base_point_ec, *_s ); encode( _public_key_A, _pk_A_ec );

    // printf( "Public key A: ");
    // for ( int _ = 0; _ < 32; _++ ) printf("%02x", _public_key_A[ _ ] ); printf("\n");

    // set prefix
    memcpy( _prefix, _ctx->hash + 32, 32 );

    // printf( "Prefix: ");
    // for ( int _ = 0; _ < 32; _++ ) printf("%02x", _prefix[ _ ] ); printf("\n");

    // printf( "s: " ); print( *_s );

    // Hash prefix || message
    sha512::cuda::Sha512_init( _ctx ); 
    sha512_multiple_args( _ctx, 32 + __message_size, 2, 32, _prefix, __message_size, __message ); sha512::cuda::Sha512_free( _ctx );

    // r
    memcpy( *_r, _ctx->hash, 64 );

    // printf( "r: " ); print( *_r );

    // Scalar multiplication r [ B ]
    set_extended_coordinates( _base_point_ec, base_point ); modular_l( *_r );
    group_element_ec _r_B { { 0 }, { 0 }, { 0 }, { 0 } }; scalar_multiplication( &_r_B, *_base_point_ec, *_r );

    // R
    encode( _R, _r_B );

    // printf( "R: ");
    // for ( int _ = 0; _ < 32; _++ ) printf("%02x", _R[ _ ] ); printf("\n");

    // Hash R || A || message
    sha512::cuda::Sha512_init( _ctx );
    sha512_multiple_args( _ctx, 32 + 32 + __message_size, 3, 32, _R, 32, _public_key_A, __message_size, __message ); sha512::cuda::Sha512_free( _ctx );

    // k
    memcpy( *_k, _ctx->hash, 64 );

    // Compute s
    modular_l( *_k ); multiplication_mod_l( *_S, *_k, *_s ); addition( *_S, *_r ); modular_l( *_S );  

    // printf( "k: " ); print( *_k );
    // printf( "S: " ); print( *_S );

    memcpy( __signature, _R, 32 ); memcpy( __signature + 32, _S, 32 );

    hipFree( _ctx ); hipFree( _base_point_ec );

    hipFree( _s ); hipFree( _r );
    hipFree( _k ); hipFree( _S );

    hipFree( _public_key_A ); hipFree( _prefix );
    hipFree( _R ); 

}

__device__ bool ed25519::cuda::verify( void* __public_key, void* __signature, void* __message, uint64_t __message_size ) {

    group_element_ec _base_point_ec { { 0 }, { 0 }, { 0 }, { 0 } };
    sha512::cuda::Sha512_Context _ctx;

    set_extended_coordinates( &_base_point_ec, base_point );

    group_element_ec _R { { 0 }, { 0 }, { 0 }, { 0 } }, _A { { 0 }, { 0 }, { 0 }, { 0 } }, _sB { { 0 }, { 0 }, { 0 }, { 0 } }, _hA { { 0 }, { 0 }, { 0 }, { 0 } }, _R_hA { { 0 }, { 0 }, { 0 }, { 0 } };
    field_element _S { 0 }, _k { 0 };

    decode( &_R, __signature ); decode( &_A, __public_key );

    memcpy( _S, __signature + 32, 32 );

    if (
        is_zero( _R.X ) ||
        is_zero( _A.X ) ||
        greather_or_equal_to( _S, l )
    ) return 0;

    sha512::cuda::Sha512_init( &_ctx ); sha512_multiple_args( &_ctx, 32 + 32 + __message_size, 3, 32, __signature, 32, __public_key, __message_size, __message ); sha512::cuda::Sha512_free( &_ctx );

    memcpy( _k, _ctx.hash, 64 ); modular_l( _k );

    scalar_multiplication( &_sB, _base_point_ec, _S ); scalar_multiplication( &_hA, _A, _k );

    point_addition( &_R_hA, _R, _hA );

    return point_equal( _sB, _R_hA );

}


